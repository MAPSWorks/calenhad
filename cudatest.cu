#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : cudatest.cu
 Author      : Martin Bucknall
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <>
#include "cudatest.h"

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
    if (err == hipSuccess)
        return;
    std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
    exit (1);
}

float *cpuReciprocal(float *data, unsigned size)
{
    float *rc = new float[size];
    for (unsigned cnt = 0; cnt < size; ++cnt) rc[cnt] = 1.0/data[cnt];
    return rc;
}


/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < vectorSize)
		data[idx] = 1.0/data[idx];
}

/**
 * Host function that copies the data and launches the work on GPU
 */
float *gpuReciprocal(unsigned size)
{
	float *rc = new float[size];
	float *data = new float[size];
	float *gpuData;

     for (unsigned i = 0; i < size; ++i) {
           data[i] = .5*(i+1);
    }

	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(float)*size));
	CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, sizeof(float)*size, hipMemcpyHostToDevice));

	static const int BLOCK_SIZE = 256;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
	reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (gpuData, size);

	CUDA_CHECK_RETURN(hipMemcpy(rc, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuData));

    float dataSum = 0.0, rcSum = 0.0;
    for (unsigned i = 0; i < size; ++i) {
        dataSum += data [i];
        rcSum += rc [i];
    }

        /* Verify the results */
                std::cout<<"dataSum = "<<dataSum<<std::endl;
        std::cout<<"rcSum = "<<rcSum<<std::endl;
    delete[] data;
	return rc;
}

